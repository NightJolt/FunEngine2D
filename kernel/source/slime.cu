#include "hip/hip_runtime.h"

#include "slime.cuh"

#include <windows.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>

#include <cstdio>





inline void check_err(hipError_t result, char const *const func, int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at line:%d code=%d(%s) \"%s\" \n", line,
			static_cast<unsigned int>(result), hipGetErrorName(result), func);
		exit(EXIT_FAILURE);
	}
}

#define checkCudaErrors(val) check_err((val), #val, __LINE__)

bool fun::cuda::slime::check_cuda_devices() {
	int32_t n = 0;

	if (hipGetDeviceCount(&n) != hipError_t::hipSuccess) {
		return false;
	}

	for (uint32_t i = 0; i < n; i++) {
		hipDeviceProp_t properties;
		checkCudaErrors(hipGetDeviceProperties(&properties, i));

		printf("#%u: %s\n", i, properties.name);
	}
    
	return n;
}





namespace {
    const uint32_t block_cnt = 1;
    const uint32_t block_dim = 1024;

    hipGraphicsResource* canvas_res;
    hipArray* canvas_arr;
}

void fun::cuda::slime::register_image(uint32_t image) {
    checkCudaErrors(hipGraphicsGLRegisterImage(&::canvas_res, image, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
}

void fun::cuda::slime::map_image() {
    checkCudaErrors(hipGraphicsMapResources(1, &::canvas_res, 0));
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&::canvas_arr, ::canvas_res, 0, 0));
}

void fun::cuda::slime::unmap_image() {
    checkCudaErrors(hipGraphicsUnmapResources(1, &::canvas_res, 0));
}




__device__ __inline__ uint32_t make_rgba(uint8_t r, uint8_t g, uint8_t b, uint8_t a) {
    return r | g << 8 | b << 16 | a << 24;
}

__device__ __inline__ uint8_t get_alpha(uint32_t color) {
    return color >> 24;
}

__device__ __inline__ uint8_t get_blue(uint32_t color) {
    return color >> 16;
}

__device__ __inline__ uint8_t get_green(uint32_t color) {
    return color >> 8;
}

__device__ __inline__ uint8_t get_red(uint32_t color) {
    return color;
}



__device__ __inline__ uint32_t coord_to_index(uint32_t x, uint32_t y, uint32_t width) {
    return x + y * width;
}


__device__ __inline__ float radians(float degrees) {
    return degrees * 0.01745f;
}


namespace {
    float* value_map;
    float* temp_value_map;

    hiprandState* curand_states;

    uint32_t canvas_size_x;
    uint32_t canvas_size_y;

    uint32_t agent_count;

    struct agent_t {
        float pos_x, pos_y;
        float angle;
    };

    agent_t* agents;

    fun::cuda::slime::agent_specs_t* agent_specs;
    fun::cuda::slime::simulation_specs_t* simulation_specs;
}

__global__ void init_curand_states(hiprandState* curand_states, uint32_t seed, uint32_t block_dim) {
    for (uint32_t i = threadIdx.x; i < block_dim; i += blockDim.x) {
        hiprand_init(seed, i, 0, &curand_states[i]);
    }
}

__global__ void init_agents(::agent_t* agents, uint32_t count, uint32_t sx, uint32_t sy, hiprandState* curand_states) {
    for (uint32_t i = threadIdx.x; i < count; i += blockDim.x) {
        agents[i].pos_x = hiprand_uniform(&curand_states[threadIdx.x]) * (sx - 1);
        agents[i].pos_y = hiprand_uniform(&curand_states[threadIdx.x]) * (sy - 1);

        agents[i].angle = hiprand_uniform(&curand_states[threadIdx.x]) * 6.283f;
    }
}

void fun::cuda::slime::init_simulation(uint32_t canvas_size_x, uint32_t canvas_size_y, uint32_t seed, uint32_t agent_count, agent_specs_t* agent_specs, simulation_specs_t* simulation_specs) {
    ::canvas_size_x = canvas_size_x;
    ::canvas_size_y = canvas_size_y;
    ::agent_count = agent_count;
    ::agent_specs = agent_specs;
    ::simulation_specs = simulation_specs;

    checkCudaErrors(hipMalloc(&::value_map, ::canvas_size_x * ::canvas_size_x * sizeof(float)));
    checkCudaErrors(hipMalloc(&::temp_value_map, ::canvas_size_x * ::canvas_size_x * sizeof(float)));
    checkCudaErrors(hipMalloc(&::curand_states, ::block_dim * sizeof hiprandState));
    checkCudaErrors(hipMalloc(&::agents, ::agent_count * sizeof ::agent_t));

    hipMemset(::value_map, 0, ::canvas_size_x * ::canvas_size_x * sizeof(float));

    init_curand_states <<<1, ::block_dim>>> (::curand_states, seed, ::block_dim);
    init_agents <<<1, ::block_dim>>> (::agents, ::agent_count, ::canvas_size_x, ::canvas_size_y, ::curand_states);
}

__global__ void step_move(float* value_map, uint32_t sx, uint32_t sy, ::agent_t* agents, uint32_t agent_count, fun::cuda::slime::agent_specs_t agent_specs, hiprandState* curand_states) {
    for (uint32_t i = threadIdx.x; i < agent_count; i += blockDim.x) {
        ::agent_t& agent = agents[i];

        float move_speed = agent_specs.move_speed;

        agent.pos_x += cos(agent.angle) * move_speed;
        agent.pos_y += sin(agent.angle) * move_speed;

        if (agent.pos_x < 0 || agent.pos_x >= sx || agent.pos_y < 0 || agent.pos_y >= sy) {
            agent.pos_x = max(0, min(sx - 1, (uint32_t)agent.pos_x));
            agent.pos_y = max(0, min(sy - 1, (uint32_t)agent.pos_y));

            agent.angle = hiprand_uniform(&curand_states[threadIdx.x]) * 6.283f;
        }

        value_map[coord_to_index((uint32_t)agents[i].pos_x, (uint32_t)agents[i].pos_y, sx)] = 1.f;
    }
}

__global__ void step_diffuse(float* value_map, float* temp_value_map, uint32_t sx, uint32_t sy, fun::cuda::slime::simulation_specs_t simulation_specs) {
    for (uint32_t i = threadIdx.x; i < sx * sy; i += blockDim.x) {
        float v = 0;
        
        for (int32_t x = -simulation_specs.diffuse_radius; x <= simulation_specs.diffuse_radius; x++) {
            for (int32_t y = -simulation_specs.diffuse_radius; y <= simulation_specs.diffuse_radius; y++) {
                int32_t nx = (i % sx) + x;
                int32_t ny = (i / sx) + y;

                if (nx >= 0 && nx < sx && ny >= 0 && ny < sy) {
                    v += value_map[coord_to_index(nx, ny, sx)];
                }
            }
        }
        
        v /= (simulation_specs.diffuse_radius * 2 + 1) * (simulation_specs.diffuse_radius * 2 + 1);

        temp_value_map[i] = value_map[i] + (v - value_map[i]) * simulation_specs.diffuse_speed;
    }
}

__global__ void step_decay(float* value_map, uint32_t sx, uint32_t sy, fun::cuda::slime::simulation_specs_t simulation_specs) {
    for (uint32_t i = threadIdx.x; i < sx * sy; i += blockDim.x) {
        value_map[i] = max(0.f, value_map[i] - simulation_specs.decay);
    }
}

__device__ float sense(float* value_map, uint32_t sx, uint32_t sy, ::agent_t& agent, fun::cuda::slime::agent_specs_t& agent_specs, float angle) {
    float v = 0;

    float sensor_angle = agent.angle + angle;

    float sensor_dir_x = cos(sensor_angle);
    float sensor_dir_y = sin(sensor_angle);

    float sensor_pos_x = agent.pos_x + sensor_dir_x * agent_specs.sensor_distance;
    float sensor_pos_y = agent.pos_y + sensor_dir_y * agent_specs.sensor_distance;

    for (int32_t x = -agent_specs.sensor_radius; x <= agent_specs.sensor_radius; x++) {
        for (int32_t y = -agent_specs.sensor_radius; y <= agent_specs.sensor_radius; y++) {
            int32_t nx = (int32_t)sensor_pos_x + x;
            int32_t ny = (int32_t)sensor_pos_y + y;

            if (nx >= 0 && nx < sx && ny >= 0 && ny < sy) {
                v += value_map[coord_to_index(nx, ny, sx)];
            }
        }
    }

    return v;
}

__global__ void step_sensor(float* value_map, uint32_t sx, uint32_t sy, ::agent_t* agents, uint32_t agent_count, fun::cuda::slime::agent_specs_t agent_specs, hiprandState* curand_states) {
    for (uint32_t i = threadIdx.x; i < agent_count; i += blockDim.x) {
        ::agent_t& agent = agents[i];

        float forward = sense(value_map, sx, sy, agent, agent_specs, 0);
        float right = sense(value_map, sx, sy, agent, agent_specs, agent_specs.sensor_angle);
        float left = sense(value_map, sx, sy, agent, agent_specs, -agent_specs.sensor_angle);

        float steer_strength = hiprand_uniform(&curand_states[threadIdx.x]);

        if (right > forward && left > forward) {
            agent.angle += (steer_strength * 2.f - 1.f) * agent_specs.turn_speed;
        } else if (left > right) {
            agent.angle -= steer_strength * agent_specs.turn_speed;
        } else if (right > left) {
            agent.angle += steer_strength * agent_specs.turn_speed;
        }
    }
}

__global__ void update_value_map(float* value_map, float* temp_value_map, uint32_t sx, uint32_t sy) {
    for (uint32_t i = threadIdx.x; i < sx * sy; i += blockDim.x) {
        value_map[i] = temp_value_map[i];
    }
}

__global__ void update_canvas(float* value_map, hipSurfaceObject_t canvas, uint32_t sx, uint32_t sy) {
    for (uint32_t i = threadIdx.x; i < sx * sy; i += blockDim.x) {
        uint8_t v = value_map[i] * 255;
        uint32_t col = make_rgba(v, v, v, 255);

        surf2Dwrite(col, canvas, i % sx * sizeof uint32_t, i / sx);
    }
}

__global__ void color_canvas(float* value_map, hipSurfaceObject_t canvas, uint32_t sx, uint32_t sy, uint8_t r, uint8_t g, uint8_t b, int32_t color_radius) {
    for (uint32_t i = threadIdx.x; i < sx * sy; i += blockDim.x) {
        float v = 0;

        for (int32_t x = -color_radius; x <= color_radius; x++) {
            for (int32_t y = -color_radius; y <= color_radius; y++) {
                int32_t nx = i % sx + x;
                int32_t ny = i / sx + y;

                if (nx >= 0 && nx < sx && ny >= 0 && ny < sy) {
                    v += value_map[coord_to_index(nx, ny, sx)];
                }
            }
        }

        const float max_val = (color_radius + .5f) * (color_radius + .5f);

        v = 1.f - abs(max_val - v) / max_val;

        uint32_t col = make_rgba(v * r, v * g, v * b, 255);
        
        surf2Dwrite(col, canvas, i % sx * sizeof uint32_t, i / sx);
    }
}

void fun::cuda::slime::step() {
    hipResourceDesc description;
	memset(&description, 0, sizeof description);
    description.resType = hipResourceTypeArray;
    description.res.array.array = ::canvas_arr;

    hipSurfaceObject_t canvas;
    checkCudaErrors(hipCreateSurfaceObject(&canvas, &description));

    step_move <<<1, ::block_dim>>> (::value_map, ::canvas_size_x, ::canvas_size_y, ::agents, ::agent_count, *::agent_specs, ::curand_states);
    step_diffuse <<<1, ::block_dim>>> (::value_map, ::temp_value_map, ::canvas_size_x, ::canvas_size_y, *::simulation_specs);
    update_value_map <<<1, ::block_dim>>> (::value_map, ::temp_value_map, ::canvas_size_x, ::canvas_size_y);
    step_decay <<<1, ::block_dim>>> (::value_map, ::canvas_size_x, ::canvas_size_y, *::simulation_specs);
    step_sensor <<<1, ::block_dim>>> (::value_map, ::canvas_size_x, ::canvas_size_y, ::agents, ::agent_count, *::agent_specs, ::curand_states);
    update_canvas <<<1, ::block_dim>>> (::value_map, canvas, ::canvas_size_x, ::canvas_size_y);
    // color_canvas <<<1, ::block_dim>>> (::value_map, canvas, ::canvas_size_x, ::canvas_size_y, ::simulation_specs->r, ::simulation_specs->g, ::simulation_specs->b, ::simulation_specs->color_radius);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipDestroySurfaceObject(canvas));
}