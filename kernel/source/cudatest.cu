#include "hip/hip_runtime.h"
#include "cudatest.cuh"

// #include <hiprand.h>
// #include <hiprand/hiprand_kernel.h>

// #include <hip/hip_runtime.h>
// #include <>
#include <windows.h>
// #include <WinGDI.h>
#include <cuda_gl_interop.h>

__global__ void cadd(int32_t* a, int32_t* b, int32_t* c) {
    *c = *a + *b;
}

int fun::cudatest::add_two_numbers(int32_t host_a, int32_t host_b) {
    int host_c;

    int* device_a;
    int* device_b;
    int* device_c;

    hipMalloc(&device_a, sizeof(int));
    hipMalloc(&device_b, sizeof(int));
    hipMalloc(&device_c, sizeof(int));

    hipMemcpy(device_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, &host_b, sizeof(int), hipMemcpyHostToDevice);

    cadd <<<1, 1>>> (device_a, device_b, device_c);

    hipMemcpy(&host_c, device_c, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return host_c;
}