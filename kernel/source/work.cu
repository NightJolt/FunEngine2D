#include "hip/hip_runtime.h"
#include "work.cuh"

#include <stdio.h>

inline void check_err(hipError_t result, char const *const func, int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at line:%d code=%d(%s) \"%s\" \n", line, static_cast<unsigned int>(result), hipGetErrorName(result), func);

		exit(EXIT_FAILURE);
	}
}

#define checkCudaErrors(val) check_err((val), #val, __LINE__)




__host__ void create_texture(color_t** texture, uint32_t width, uint32_t height) {
    checkCudaErrors(hipMalloc(texture, width * height * sizeof(color_t)));
}

__host__ void free_texture(color_t* texture) {
    checkCudaErrors(hipFree(texture));
}





__global__ void cuda_fill_texture(color_t* texture, uint32_t width, uint32_t height, color_t color) {
    uint32_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;
    const uint32_t size = width * height;
    
    for (uint32_t i = ind; i < size; i += stride) {
        texture[i] = color;
    }
}

__host__ void fill_texture(color_t* texture, uint32_t width, uint32_t height, color_t color) {
    cuda_fill_texture <<<256, 256>>> (texture, width, height, color);
}



void create_surface(color_t** surface, uint32_t width, uint32_t height) {
    *surface = (color_t*)malloc(width * height * sizeof(color_t));
}

void free_surface(color_t* surface) {
    free(surface);
}




__host__ void texture_to_surface(color_t* surface, color_t* texture, uint32_t width, uint32_t height) {
    checkCudaErrors(hipMemcpy(surface, texture, width * height * sizeof(color_t), hipMemcpyDeviceToHost));
}

__host__ void surface_to_texture(color_t* texture, color_t* surface, uint32_t width, uint32_t height) {
    checkCudaErrors(hipMemcpy(texture, surface, width * height * sizeof(color_t), hipMemcpyHostToDevice));
}



__global__ void cuda_blit_texture(
    color_t* target_texture,
    color_t* source_texture,
    channel_t* mask,
    uint32_t target_x,
    uint32_t target_y,
    uint32_t target_w,
    uint32_t target_h,
    uint32_t source_x,
    uint32_t source_y,
    uint32_t source_w,
    uint32_t source_h
) {
    uint32_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;
    const uint32_t size = target_w * target_h;
    
    for (uint32_t i = ind; i < size; i += stride) {
        uint32_t tx = i % target_w;
        uint32_t ty = i / target_w;
        
        uint32_t sx = (float)(tx - target_x) / target_w * source_w + source_x;
        uint32_t sy = (float)(ty - target_y) / target_h * source_h + source_h;

        uint32_t source_ind = sy * source_w + sx;
        
        if (mask && mask[source_ind] == 0) {
            continue;
        }
    }
}

__host__ void blit_texture(
    color_t* target_texture,
    color_t* source_texture,
    channel_t* mask,
    uint32_t target_x,
    uint32_t target_y,
    uint32_t target_w,
    uint32_t target_h,
    uint32_t source_x,
    uint32_t source_y,
    uint32_t source_w,
    uint32_t source_h
) {
    cuda_blit_texture <<<256, 256>>> (
        target_texture,
        source_texture,
        mask,
        target_x,
        target_y,
        target_w,
        target_h,
        source_x,
        source_y,
        source_w,
        source_h
    );
}