#include "hip/hip_runtime.h"
#include "cudatest.cuh"

__global__ void cadd(int* a, int* b, int* c) {
    *c = *a + *b;
}

int fun::CudaTesting::AddTwoNumbers(int host_a, int host_b) {
    int host_c;

    int* device_a;
    int* device_b;
    int* device_c;

    hipMalloc(&device_a, sizeof(int));
    hipMalloc(&device_b, sizeof(int));
    hipMalloc(&device_c, sizeof(int));

    hipMemcpy(device_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, &host_b, sizeof(int), hipMemcpyHostToDevice);

    cadd <<<1, 1>>> (device_a, device_b, device_c);

    hipMemcpy(&host_c, device_c, sizeof(int), hipMemcpyDeviceToHost);

    return host_c;
}